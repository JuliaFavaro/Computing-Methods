#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void) { //this will be executed in the GPU, DEVICE
  printf("Hello World from GPU! (block: %d thread: %d)\n",blockIdx.x,threadIdx.x); //built in variables of the Cuda programming
}

int main(void) { //this will be executed in the PC, HOST
  mykernel <<<3,4>>>(); //launch KERNEL (with the number of blocks and threads we want to use. The numbering starts from 0 up to n-1
  hipDeviceSynchronize(); //Kernel is asynchronous by default. You are asking to wait for the results of the kernel before moving to the next line
  printf("Hello World from Host!\n");
  return 0;
}
// it will show on the screen the execution in parallel by different threads in different blocks. The order of printing is different in every run (like in normal parallel programming) 
//you cannot assume to use the results from one threads to be entry data for another threads because you will never know the order of execution
// if you comment the cudaDEviceSynchronize you will only see on screen the "Hello world from host" line
