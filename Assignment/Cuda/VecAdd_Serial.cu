//how you would write this problem in your standard CPU (PC)

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1048576

void RandomVector(int *a, int nn){
  for (int i=0;i<nn;i++) {
    a[i]=rand()%100+1; 
  }
}

//serial sum
void VecAddSerial(int *a, int *b, int *c){ //pointers to two arrays
  for (int i=0;i<N;i++){
    c[i] = a[i]+b[i];
  }
}

int main(void) {
  int *h_a, *h_b, *h_c;
  int size = N*sizeof(int);

  float time;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //Alloc in Host (and filling)
  h_a = (int *)malloc(size); //dynamical allocation of memory
  h_b = (int *)malloc(size);
  h_c = (int *)malloc(size);
  RandomVector(h_a,N); //you fill them with random numbers
  RandomVector(h_b,N);

  //start time
  hipEventRecord(start);

  //Launch Serial Sum on CPU
  VecAddSerial(h_a,h_b,h_c); //if you give the name of an array you are sending the pointer to the first element of the array

 //stop time
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  //Print Result
  //  for(int i=0;i<N;i++){
  //  printf ("%d) h_a:%d h_b:%d h_c:%d\n",i,h_a[i],h_b[i],h_c[i]);
  //}

  //print time
  printf("Time: %3.5f ms\n",time);

  //Cleanup
  free(h_a);
  free(h_b);
  free(h_c);

  return(0);
}
//it will print only the execution time in Serial Programming
