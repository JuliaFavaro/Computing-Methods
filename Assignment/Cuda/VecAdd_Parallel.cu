
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1048576
#define THREADS_PER_BLOCK 128

void RandomVector(int *a, int nn){ //the kernel should be adapted with an index to an architecture where the total number of threads is equal to the number of vectors you are actually summing
  for (int i=0;i<nn;i++) {
    a[i]=rand()%100+1; 
  }
}

//kernel
__global__ void VecAddGpu(int *a, int *b, int *c){
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  c[index] = a[index]+b[index];
}

int main(void) {
  int *h_a, *h_b, *h_c;
  int *d_a, *d_b, *d_c;
  int size = N*sizeof(int);

  float time;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //Alloc in Host (and filling)
  h_a = (int *)malloc(size); //you should always starts pointers in host with a h_name
  h_b = (int *)malloc(size);
  h_c = (int *)malloc(size); // where you want results
  RandomVector(h_a,N);
  RandomVector(h_b,N);

  //Alloc in Device
  hipMalloc((void **)&d_a, size); //the pointers are defined in a different space than the memory of the host, d_name
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  //Copy input vectors form host to device
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

  //start time
  hipEventRecord(start);

  //Launch Kernel  on GPU
  VecAddGpu<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a,d_b,d_c); // if you want to optimize you need to ask for the minumum number of blocks necessary 
  hipDeviceSynchronize();

  //stop time
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  hipError_t KernelError=hipGetLastError(); // you should check for errors
  printf("Error %s\n",hipGetErrorString(KernelError));

  //Copy back the results
  hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost); //opposite direction

  //Print Result
  //  for(int i=0;i<N;i++){
  //  printf ("%d) h_a:%d h_b:%d h_c:%d\n",i,h_a[i],h_b[i],h_c[i]);
  //}

  //print time
  printf("Time: %3.5f ms\n",time);

  //Cleanup. always necessary for dynamical memory
  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return(0);
}

//if you execute N blocks with 1 thread : it will take way more than time the serial programm! Only 2 blocks can be executed at the same time: you are basically only executing 2 threads at a time
//if you execute 1 block with N threads: it will take way less time (a factor of 1000)!! You can only go up to 1024 with our GPU MODEL otherways you are taking more resources than available
// this is why you should know the caracteristics of your videocard for maximum explotation-> you should actually check if the division is an integer I guess
